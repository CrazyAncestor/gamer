#include "CUAPI.h"

#if ( defined GPU  &&  defined SUPPORT_GRACKLE )




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemAllocate_Grackle
// Description :  Allocate GPU and CPU memory for the Grackle solver
//
// Parameter   :  Che_NPG : Number of patch groups evaluated simultaneously by GPU
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemAllocate_Grackle( const int Che_NPG )
{

// nothing to do if Grackle is disabled
   if ( !GRACKLE_ACTIVATE )   return;


// size of the global memory array(s)
   const long Che_MemSize_In   = sizeof(real)*Che_NPG*CHE_NIN  *CUBE(PS2);
   const long Che_MemSize_Prep = sizeof(real)*Che_NPG*CHE_NPREP*CUBE(PS2);

// output the total memory requirement
   long TotalSize = Che_MemSize_In;

   if ( MPI_Rank == 0 )
      Aux_Message( stdout, "NOTE : total memory requirement in GPU Grackle solver = %ld MB\n", TotalSize/(1<<20) );


// allocate the device memory
// --> not necessary for now


// allocate the host memory by CUDA
   for (int t=0; t<2; t++)
   {
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Che_Array[t], Che_MemSize_Prep )  );
   }

} // FUNCTION : CUAPI_MemAllocate_Grackle



#endif // #if ( defined GPU  &&  defined SUPPORT_GRACKLE )
