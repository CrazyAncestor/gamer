#include "CUAPI.h"

#if ( defined GPU  &&  defined SUPPORT_GRACKLE )




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_Grackle
// Description :  Free the device and host memory previously allocated by CUAPI_MemAllocate_Grackle()
//
// Parameter   :  None
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_Grackle()
{

// free the device memory
// --> not necessary for now


// free the host memory allocated by CUDA
   for (int t=0; t<2; t++)
   {
      if ( h_Che_Array[t] != NULL )    CUDA_CHECK_ERROR(  hipHostFree( h_Che_Array[t] )  );

      h_Che_Array[t] = NULL;
   }

} // FUNCTION : CUAPI_MemFree_Grackle



#endif // #if ( defined GPU  &&  defined SUPPORT_GRACKLE )
